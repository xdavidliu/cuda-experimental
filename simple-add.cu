
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>  // std::exit

void check(const hipError_t err, const char *msg = "unknown") {
    if (err == hipSuccess) {
        std::printf("succeeded %s\n", msg);
    } else {
        std::fprintf(stderr, "failed %s with err = %s\n",
                     msg, hipGetErrorString(err));
        std::exit(1);
    }
}

template <typename T> class DevPtr {  // "device pointer"
    T *ptr;
    public:
    DevPtr() : ptr(0) {
        check(hipMalloc(&ptr, sizeof *ptr), "construct");
    }
    T *get() const {
        return ptr;
    }
    const T value() const {
        T val;
        check(hipMemcpy(&val, get(), sizeof val, hipMemcpyDeviceToHost), "value");
        return val;
    }
    ~DevPtr() {
        check(hipFree(ptr), "destruct");
    }
};

__global__ void add(int *c, const int a, const int b) {
    *c = a + b;
}

int main() {
    DevPtr<int> dp;
    add<<<1,1>>>(dp.get(), 5, 7);
    std::printf("foo %d\n", dp.value());
}
